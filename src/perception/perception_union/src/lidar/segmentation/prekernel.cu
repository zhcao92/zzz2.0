#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "lidar/segmentation/prekernel.h"

constexpr double EPS = 1e-6;
constexpr double K_CV_PI = 3.1415926535897932384626433832795;

__device__ int dev_F2I(float val, float ori, float scale){
  return __float2int_rd((ori - val) * scale);
}

__device__ float dev_Pixel2Pc(int in_pixel, float in_size, float out_range)
{
	float res = 2.0f * out_range / in_size ;
	return out_range - (static_cast<float>(in_pixel) + 0.5f) * res;
}

__global__ void mathmatical_cal(unsigned int ch_size, float* dev_direction_data, 
								float* dev_distance_data, float dev_range, 
								int dev_width, int dev_height)
{
	for(unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < (ch_size); i+= blockDim.x * gridDim.x){
			int row = blockIdx.x;
			int col = threadIdx.x;
			float center_x = dev_Pixel2Pc(row, dev_height, dev_range);
			float center_y = dev_Pixel2Pc(col, dev_width, dev_range);
			dev_direction_data[i] = static_cast<float>(atan2(center_y, center_x) / (2.0 * K_CV_PI));
			dev_distance_data[i] = static_cast<float>(hypot(center_x, center_y) / dev_range - 0.5);

	}
}

__global__ void cuda_FLT2INT(float* in, int* out, int ch_size)
{
	for(unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
		index < (ch_size); index += blockDim.x * gridDim.x) {
			out[index] = static_cast<int>((in[index]));
		}
}

__global__ void cloud_compute_average(float* dev_max_height_data_, float* dev_mean_height_data_,
		float* dev_count_data_, float* dev_mean_intensity_data_,
		float* dev_nonempty_data_, unsigned int ch_size, float* dev_log_table_ ) {       
	for(unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; 
			index < (ch_size); index += blockDim.x * gridDim.x) {
		float count_data_ = dev_count_data_[index];
		float mean_height_data_ = dev_mean_height_data_[index];
		float mean_intensity_data_ = dev_mean_intensity_data_[index];
		///> The ESP value can be changed 
		if (count_data_ < EPS) {  
			dev_max_height_data_[index] = 0.0f;
		} else {
			mean_height_data_ /= count_data_;
			mean_intensity_data_ /= count_data_;
			dev_nonempty_data_[index] = 1.0f;
		}
		int count = (int)(count_data_);
		count_data_ = dev_log_table_[count];
		dev_count_data_[index] = count_data_;
		dev_mean_intensity_data_[index] = mean_intensity_data_;
		dev_mean_height_data_[index] = mean_height_data_;
	}
}



__global__ void scatter_point2grid(const PointXYZI* dev_pc, unsigned int pc_size, int* writing_location, int* point_id, float dev_range, int dev_width,
									int dev_height, float dev_max_h, float dev_min_h, float dev_inv_res_x, float dev_inv_res_y, int ch_size){
	for(unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < (pc_size); i+= blockDim.x * gridDim.x){
			PointXYZI ptc = dev_pc[i];
			int dev_px = dev_F2I(ptc.y, dev_range, dev_inv_res_x);  // col
			int dev_py = dev_F2I(ptc.x, dev_range, dev_inv_res_y);  // row

			if (ptc.z <= dev_min_h || ptc.z >= dev_max_h || dev_px >= dev_width 
				|| dev_px < 0 || dev_py >= dev_height || dev_py < 0) {
					continue;
			}


			int idx = dev_py * dev_width + dev_px;

			int loc =  atomicAdd(&writing_location[idx], 1);
	
			point_id[loc] = i;
		}
	}

__global__ void find_maximum(const PointXYZI* dev_pc, int* writing_location, int* point_id, float* dev_max_height, float* dev_top_intensity,
							float min_height, int ch_size)
{
	for(unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < (ch_size); i += blockDim.x * gridDim.x){
			float max_height = min_height;
			float top_intensity = 0.0f;
			
		for(int j = writing_location[i]; j < writing_location[i + 1]; j++){
			PointXYZI ptc = dev_pc[point_id[j]];
			float itensity = ptc.i / 255.0f;
			float height = ptc.z;
			if(max_height < height){
				max_height = height;
				top_intensity = itensity;
			}
		
		}

		dev_max_height[i] = max_height;
		dev_top_intensity[i] = top_intensity;
		

	}
}

__global__ void cloud_accumulation(const PointXYZI* dev_pc, unsigned int pc_size, float* dev_mean_height_data_, 
								float* dev_mean_intensity_data_, float* dev_count_data_, float dev_range, int dev_width,
								int dev_height, float dev_max_h, float dev_min_h, float dev_inv_res_x,
								float dev_inv_res_y, int ch_size){
	for(unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
			i < (pc_size); i+= blockDim.x * gridDim.x){
		PointXYZI ptc = dev_pc[i];
		int dev_px = dev_F2I(ptc.y, dev_range, dev_inv_res_x);  // col
		int dev_py = dev_F2I(ptc.x, dev_range, dev_inv_res_y);  // row
		if (ptc.z <= dev_min_h || ptc.z >= dev_max_h || dev_px >= dev_width 
				|| dev_px < 0 || dev_py >= dev_height || dev_py < 0) {
			continue;
		}
		int idx = dev_py * dev_width + dev_px;
		float pz = ptc.z;
		float pi = ptc.i / 255.0f;
		atomicAdd(&dev_mean_height_data_[idx], pz);
		atomicAdd(&dev_mean_intensity_data_[idx], pi);
		atomicAdd(&dev_count_data_[idx], 1.0f);
	}
}
