#include "hip/hip_runtime.h"
/*
 * Copyright 2020-2022 Novauto. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "lidar/segmentation/feature_generator_cuda.h"
#include "lidar/segmentation/prekernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <memory.h>
#include <cmath>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
// #include <pcl/point_types.h>
// #include <pcl/point_cloud.h>
#include <stdio.h>
#include <assert.h>
 
int BLOCK_X = 1024;

int Get_Block(int N)
{
  return (N + BLOCK_X - 1) / BLOCK_X;
} 
 // Convenience function for checking CUDA runtime API results
 // can be wrapped around any runtime API call. No-op in release builds.
 inline hipError_t checkCuda(hipError_t result) {
 #if defined(DEBUG) || defined(_DEBUG)
   if (result != hipSuccess) {
     fprintf(stderr, "CUDA Runtime Error: %s\n",
       hipGetErrorString(result));
     assert(result == hipSuccess);
   }
 #endif
   return result;
 }

 void ExclusiveScan(int *input, int ele_num)
{
  thrust::device_ptr<int> dev_ptr(input);

  thrust::exclusive_scan(dev_ptr, dev_ptr + ele_num, dev_ptr);
  checkCuda(hipDeviceSynchronize());

}

FeatureGenerator::~FeatureGenerator(){
    hipFree(dev_log_table_);
    hipFree(writing_location_);
    hipFree(starting_points_id_);
}

void FeatureGenerator::free_out_feature_blocb()
{
 hipFree(dev_out_blob_);
}
 

bool FeatureGenerator::init(void* out_blob, 
    int height = 640, int width = 640, float range = 60.0f, 
    float min_height = -5.0f, float max_height = 5.0f, 
    bool intensity_normalized = false, int batchsize = 1, 
    int channel = 8) {

  dev_out_blob_ = (float*)out_blob;
  width_   = width;
  height_  = height;
  channel_ = channel;  
  batch_   = batchsize;
  range_   = range;
  min_height_   = min_height;
  max_height_   = max_height;
  intensity_normalized_ = intensity_normalized; 
  channel_size_         = height_ * width_;
  channel_bytes_size_   = channel_size_ * sizeof(float);

  max_height_data_ = (float*)malloc(channel_bytes_size_);
  top_intensity_data_ = (float*)malloc(channel_bytes_size_);

  size_t log_table_bytes    = LOG_TABLE_SIZE * sizeof(float);
  checkCuda(hipMalloc((void **)&dev_log_table_, log_table_bytes));
  checkCuda(hipMalloc((void **)&writing_location_, sizeof(int) * (channel_size_ + 1)));
  checkCuda(hipMalloc((void **)&starting_points_id_, sizeof(int) * (channel_size_ + 1)));

  int channel_index = 0;

  dev_max_height_data_     = dev_out_blob_ + channel_size_ * channel_index++;
  dev_mean_height_data_    = dev_out_blob_ + channel_size_ * channel_index++;
  dev_count_data_          = dev_out_blob_ + channel_size_ * channel_index++;
  dev_direction_data_      = dev_out_blob_ + channel_size_ * channel_index++;
  dev_top_intensity_data_  = dev_out_blob_ + channel_size_ * channel_index++;
  dev_mean_intensity_data_ = dev_out_blob_ + channel_size_ * channel_index++;
  dev_distance_data_       = dev_out_blob_ + channel_size_ * channel_index++;
  dev_nonempty_data_       = dev_out_blob_ + channel_size_ * channel_index++;

  float* log_table_;
  
  checkCuda( hipHostMalloc((void**)&log_table_, log_table_bytes) );

  for (int i = 0; i < LOG_TABLE_SIZE; ++i) {
    log_table_[i] = std::log1p((float)i);
  }



  checkCuda( hipMemcpy(dev_log_table_, log_table_, 
      log_table_bytes, hipMemcpyHostToDevice) );

  hipHostFree(log_table_);      

  return true;
}


void FeatureGenerator::generate(const void* pointcloud_ptr, int num) {
    
     PointXYZI* dev_pc;
     unsigned int pc_size = num;
     size_t point_bytes_size = num * sizeof(PointXYZI);
     PointXYZI* pc_ptr = (PointXYZI*)pointcloud_ptr;
     
     float inv_res_x = 0.5f * static_cast<float>(width_) / (range_);
     float inv_res_y = 0.5f * static_cast<float>(height_) / (range_);

     checkCuda( hipMalloc((void**)&dev_pc, point_bytes_size) );
     checkCuda(hipMalloc((void**)&point_id_, sizeof(int) * pc_size));

     checkCuda( hipMemsetAsync((void*)dev_mean_height_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_count_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_top_intensity_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_mean_intensity_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_nonempty_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_direction_data_, 0, channel_bytes_size_) ); 
     checkCuda( hipMemsetAsync((void*)dev_distance_data_, 0, channel_bytes_size_) ); 

     checkCuda( hipMemsetAsync((void**)writing_location_, 0, sizeof(int) * (channel_size_ + 1)));
     checkCuda( hipMemsetAsync((void**)starting_points_id_, 0, sizeof(int) * (channel_size_ + 1)));
 


     hipStream_t stream1, stream2;

     hipStreamCreate(&stream1);
     hipStreamCreate(&stream2);

     mathmatical_cal<<<height_, width_, 0, stream1>>>(channel_size_, dev_direction_data_,dev_distance_data_,
                                                            range_, width_, height_);
 
     checkCuda( hipMemcpyAsync((void*)dev_pc, (void*)pc_ptr, point_bytes_size, hipMemcpyHostToDevice, stream2));
 
     cloud_accumulation<<<Get_Block(pc_size), BLOCK_X, 0, stream2>>>(dev_pc, pc_size, dev_mean_height_data_,
                                                      dev_mean_intensity_data_, dev_count_data_, range_, 
                                                      width_, height_, max_height_, min_height_, inv_res_x, inv_res_y, channel_size_);

     cuda_FLT2INT<<<Get_Block(channel_size_), BLOCK_X, 0, stream2>>>(dev_count_data_, writing_location_, channel_size_);

     checkCuda(hipStreamSynchronize(stream2));
 
     ExclusiveScan(writing_location_, channel_size_ + 1);

     checkCuda(hipMemcpy(starting_points_id_, writing_location_, sizeof(int) * (channel_size_ + 1), hipMemcpyDeviceToDevice));

     scatter_point2grid<<<Get_Block(pc_size), BLOCK_X>>>(dev_pc, pc_size, writing_location_, point_id_, range_, width_, height_, 
                                                          max_height_, min_height_, inv_res_x, inv_res_y, channel_size_);
                                                
     find_maximum<<<Get_Block(channel_size_), BLOCK_X>>>(dev_pc, starting_points_id_, point_id_, dev_max_height_data_, 
                                                        dev_top_intensity_data_, min_height_, channel_size_);

     cloud_compute_average<<<Get_Block(channel_size_), BLOCK_X>>>(dev_max_height_data_, dev_mean_height_data_, dev_count_data_,
                          dev_mean_intensity_data_, dev_nonempty_data_, channel_size_, dev_log_table_);

     checkCuda(hipStreamSynchronize(stream2));
     checkCuda(hipStreamSynchronize(stream1));

     hipFree(dev_pc);
     hipFree(point_id_);

     hipStreamDestroy(stream1);
     hipStreamDestroy(stream2);
    
   
}
 